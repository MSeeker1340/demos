#include "hip/hip_runtime.h"
__global__ void matmul({real} *M, {real} *N, {real} *P, int Width)
{{
    // Compute M * N and store result in P
    // M and N are Width * Width matrices
    __shared__ {real} Ms[{TW}][{TW}];
    __shared__ {real} Ns[{TW}][{TW}];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = blockIdx.y * {TW} + ty;
    int Col = blockIdx.x * {TW} + tx;

    {real} Pvalue = {fzero};
    for (int ph = 0; ph < ceil(Width / ({real}){TW}); ++ph)
    {{
        // Cooperatively load tile into shared memory
        if (Row < Width && ph*{TW} + tx < Width)
        {{
            Ms[ty][tx] = M[Row*Width + ph*{TW} + tx];
        }}
        else
        {{
            Ms[ty][tx] = {fzero};
        }}
        if (Col < Width && ph*{TW} + ty < Width)
        {{
            Ns[ty][tx] = N[(ph*{TW} + ty)*Width + Col];
        }}
        else
        {{
            Ns[ty][tx] = {fzero};
        }}
        __syncthreads();

        {loop}
        __syncthreads();
    }}

    if (Row < Width && Col < Width)
    {{
        P[Row*Width + Col] = Pvalue;
    }}
}}
